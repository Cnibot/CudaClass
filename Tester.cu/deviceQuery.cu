#include <iostream>

int main() {
    int devices;

    hipGetDeviceCount(&devices);
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);

    printf("  Device name: %s\n", prop.name);
    printf("  Memory Clock Rate (KHz): %d\n", prop.memoryClockRate);
    printf("  Memory Bus Width (bits): %d\n", prop.memoryBusWidth);
    printf("  Peak Memory Bandwidth (GB/s): %f\n\n", 2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);

    getchar();
}
