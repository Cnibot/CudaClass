
#include <hip/hip_runtime.h>
#include <iostream>

#define N 10000

float *a_cpu, *b_cpu, *c_cpu;
float *a_gpu, *b_gpu, *c_gpu;

__global__ void add(float *a_gpu, float *b_gpu, float *c_gpu, int n){
    for(int i = 0; i < n; i++){
        c_gpu[i] = a_gpu[i] + b_gpu[i];
    }
}

int main() {

    a_cpu = (float*)malloc(sizeof(float)*N);
    b_cpu = (float*)malloc(sizeof(float)*N);
    c_cpu = (float*)malloc(sizeof(float)*N);

    hipMalloc((float**)&a_gpu, sizeof(float) * N);
    hipMalloc((float**)&b_gpu, sizeof(float) * N);
    hipMalloc((float**)&c_gpu, sizeof(float) * N);

    for (int i = 1; i <= N; i++){
        a_cpu[i] = -i;
        b_cpu[i] = i*2;
        c_cpu[i] = a_cpu[i] + b_cpu[i];
    }

    hipMemcpy(a_gpu, a_cpu, sizeof(float) * N, hipMemcpyHostToDevice);
    hipMemcpy(b_gpu, b_cpu, sizeof(float) * N, hipMemcpyHostToDevice);
    hipMemcpy(c_gpu, c_cpu, sizeof(float) * N, hipMemcpyHostToDevice);

    add<<<1,10>>>(a_gpu, b_gpu, c_gpu, N);

    hipMemcpy(c_cpu, c_gpu, sizeof(float) * N, hipMemcpyDeviceToHost);

    for (int i = 1; i <= N; i++){
        printf("%.0f\n", c_cpu[i]);
    }

    hipFree(a_gpu);
    hipFree(b_gpu);
    hipFree(c_gpu);
    free(a_cpu);
    free(b_cpu);
    free(c_cpu);

    getchar();
    return 0;
}